//--Includes--------------------------------------------------------------------
#include "DTW.cuh"

#include <iostream>
#include <vector>
#include <fstream>
#include <sstream>
#include <string.h>

using namespace dtw; // DTW.cuh

//--Main------------------------------------------------------------------------
int main(int argc, char **argv) {
    std::cout << "Execution starting..." << std::endl;

	//--Parse arguments---------------------------------------------------------
	std::cout << "Parsing parameters..." << std::endl;

    std::string filename;

    if (argc != 2) {
        std::cout << "ERROR: Wrong parameters" << std::endl;
        std::cout << "Usage: ./exe NW.seq" << std::endl;
    } 
    else {
        filename = argv[1];
    }


    //--Files managment---------------------------------------------------------
	std::cout << "Reading files..." << std::endl;

    // Struct to hold all the required data (see NW.cuh)
    DataStruct sequencesData; 
    initializeDataStruct(&sequencesData);

    // File reading
    std::ifstream file;
    file.open(filename, std::fstream::in);
    if (file.is_open()) {
        /*	NOTE:
		    - In our case we use .seq files so we use one file with all the 
              data,line starts with < or > to distinguish the pair.
            - Note that we DO NOT want the 1st char in the line.
        */

        // Line reading
        std::string line; 
        while(getline(file, line)) { // Read line

            // Auxiliars
            std::stringstream lineStream;
            std::vector<uint32_t> signal;
            uint32_t sample;

            switch (line[0]) { // Look at first character
                case '>':
                    line.erase(0, 1); // Erase first character from line
                    lineStream.str(line); // String to stream
                    while(lineStream >> sample) { // Read stream
                        signal.push_back(sample);
                    }
                    storeTarget(&(signal[0]), signal.size(), &sequencesData);
                    break;
                case '<':
                    line.erase(0, 1); // Erase first character from line
                    lineStream.str(line); // String to stream
                    while(lineStream >> sample) { // Read stream
                        signal.push_back(sample); 
                    }
                    storeQuery(&(signal[0]), signal.size(), &sequencesData);
                    break;
                default:
                    std::cout << "ERROR: problem reading files" << std::endl;
                    return -1;
            }
        }
        file.close();
    } 
    else {
        std::cout << "ERROR: File " << filename << " not found." << std::endl;
        return -1;
    }

    /*
    //TEST CODE START
        std::cout << "Total number of targets/querys: " << sequencesData.nTargets << " / "
            << sequencesData.nQuerys
            << "\nTotal elements in targets/querys: "
            << sequencesData.targetRefs[sequencesData.nTargets] << " / "
            << sequencesData.queryRefs[sequencesData.nQuerys]<< std::endl;
        
        for(int i = 0; i < sequencesData.nTargets; i++) {
            std::cout << "\nTarget "<< i+1 << ": (reference " << sequencesData.targetRefs[i] << ")\n";
            for (int t = sequencesData.targetRefs[i]; t < sequencesData.targetRefs[i+1]; t++) {
                std::cout << sequencesData.targets[t] << " ";
            }

            std::cout << "\nQuery "<< i+1 << ": (reference " << sequencesData.queryRefs[i] << ")\n";
            for (int q = sequencesData.queryRefs[i]; q < sequencesData.queryRefs[i+1]; q++) {
                std::cout << sequencesData.querys[q] << " ";
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
    //TEST CODE END 
    */


    //--Parameter definitions---------------------------------------------------
    std::cout << "Defining parameters..." << std::endl;

    // General parameters
    /*  NOTE:
        - For benchmarking purposes we want a certain amount of alignments that
          keep the work (cells computed) constant.
        - In our case we will only have 1 target and 1 query and align them
          several times. 
    */
    int nAlignments = 12000000000 /
        (sequencesData.targetRefs[1] * sequencesData.queryRefs[1]);

    // Kernel launching parameters
    int nThreadsPerBlock = 32; //TODO: parameterize
    int nBlocks = nAlignments/nThreadsPerBlock;
    if(nAlignments % nThreadsPerBlock) {
        nBlocks++;
    }


    //--Set GPU environment up--------------------------------------------------
    std::cout << "Preparing GPU..." << std::endl;

    /*  NOTE:
        - Pointers in GPU structures are DEVICE pointers, used only in GPU.
        - sequencesDataGPU does NOT need to be initialized, all pointers are 
          initialized during the copy to the GPU.
        - When passing a struct to the kernel a deep copy is made.
    */

    // Allocate and initialize results struct in GPU
    Results resultsGPU;
    initializeResultsGPU(&resultsGPU, nAlignments);

    // Set matrixesData
    MatrixesData matrixesGPU;
    setMatrixesDataGPU(&matrixesGPU, &sequencesData, nAlignments);

    // Copy data required to perform the alignments
    DataStruct sequencesDataGPU;
    copySequencesToGPU(&sequencesData, &sequencesDataGPU);


    //--DTW alignment-----------------------------------------------------------
    std::cout << "Launching kernels...\n\tnBlocks = " << nBlocks 
        << "\tnThreadsPerBlock = " << nThreadsPerBlock << std::endl;

    // Create and start counter
    TIMER_CREATE;
    TIMER_START;

    // Perform aligments in GPU
    dtw_align_gpu<<<nBlocks, nThreadsPerBlock>>>(
        sequencesDataGPU, 
        matrixesGPU,
        resultsGPU);
    
    // Stop counter and get time
    TIMER_STOP; 
    TIME_ELAPSED(kernelTime);

    // Check for error
    hipDeviceSynchronize();
    CHECK_LAST_CUDA_ERROR();


    //--Recover data from GPU---------------------------------------------------
    std::cout << "Recovering results..." << std::endl;

    // Copy results to CPU
    Results resultsCPU;
    copyResultsToCPU(&resultsCPU, &resultsGPU, nAlignments);


    //--Print results-----------------------------------------------------------
    // Compute values
    const double seconds = kernelTime / 1000;
    const double gCells = ((double) sequencesData.queryRefs[1] 
        * sequencesData.targetRefs[1] * nAlignments) / 1000000000L;
    
    // Performance info
    std::cout << "Performance results:"
        << "\n\tTotal number of alignments: " << nAlignments
        << "\n\tTotal number of cells (GCells): " << gCells
        << "\n\tKernel execution time (s):  " << seconds
        << "\n\tGCells/s: " << gCells / seconds
        << std::endl;

    // Alignments info
    /*
    std::cout << "Alignment results:";
    for(int i = 0; i < nAlignments; i++) {
        std::cout << "\n\tAlIdx: " << i 
            << "\n\tScore: " << resultsCPU.results[i] 
            << std::endl;
    }
    std::cout << std::endl;
    */


    //--Freeing memory----------------------------------------------------------
    std::cout << "Freeing memory..." << std::endl;

    freeStructsCPU(&sequencesData, &resultsCPU);
    freeStructsGPU(&sequencesDataGPU, &resultsGPU);


    return 0;
}
